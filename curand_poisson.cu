#include "hip/hip_runtime.h"
// See https://docs.nvidia.com/cuda/hiprand/device-api-overview.html#poisson-api-example

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

__global__ void setup_kernel(hiprandState *state) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_draws;
    i += blockDim.x * gridDim.x) {
    /* Each thread gets same seed, a different sequence
        number, no offset */
      hiprand_init(1234, i, 0, &state[i]);
    }
}

__global__ void simple_device_API_kernel(hiprandState *state,
                    float *draws, const long n_draws, const int draw_per_thread) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_draws; i += blockDim.x * gridDim.x) {
    hiprandState localState = state[i];
    /* Copy state to local memory for efficiency */
    float draw = 0;
    for (int j = 0; j < draw_per_thread; ++j) {
        draw += hiprand_poisson(&localState, j);
    }
    draws[i] = draw;
    /* Copy state back to global memory */
    state[i] = localState;
  }
}

int main() {
  hiprandState *devStates;

  const long total_draws = 1 << 20;
  const int draw_per_thread = 128;

  float* draws;
  CUDA_CALL(hipMalloc((void**)&draws, total_draws * sizeof(float)));
  CUDA_CALL(hipMalloc((void **)&devStates, total_draws *
              sizeof(hiprandState)));

  const size_t blockSize = 64;
  const size_t blockCount = (total_draws + setup_blockSize - 1) / setup_blockSize;
  setup_kernel<<<blockCount, blockSize>>>(devStates);

  simple_device_API_kernel<<<blockSize, blockCount>>>(devStates, draws, total_draws, draw_per_thread);

  CUDA_CALL(hipFree(draws));
  CUDA_CALL(hipFree(devResults));
}